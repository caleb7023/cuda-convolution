
// author: caleb7023

#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void convolution1d_channel(float *input, float *kernel, float *output, int ic, int kc, int is, int ks, int s, int p, int os){
    
    /**
     * input: input image
     * kernel: kernel
     * output: output image
     * ic: input channels
     * kc: kernel channels
     * is: input size
     * ks: kernel size
     * s: stride
     * p: padding
     * os: output size
     */
    
    int  k_ = threadIdx.x; // kernel position
    int ic_ = threadIdx.y; // input channel
    int  o_ = blockIdx.x; // target position in the output
    int kc_ = blockIdx.y; // kernel/output channel

    int i = o_-p + k_*s; // input position

    if (kc_ < kc && // kernel channel
         o_ < os && // target position in the input
        ic_ < ic && // input channel
         k_ < ks && // kernel position
        0<=i && i<is)// check if the input position is valid
    {
        atomicAdd(&output[kc_*os + o_], input[ic_*is + i] * kernel[kc_*ks + k_]);
    }
}



/**
 *
 * @brief Performs a convolution operation on the input 1D array.
 *
 * The input size does not include the channel size
 * The kernel size does not include the channel size too
 * input, output and kernels are 1D arrays
 *
 *
 * @param input The input array.
 * @param output The output array.
 * @param kernels The convolution kernels.
 * @param input_channels The number of channels in the input array.
 * @param kernel_channels The number of channels in the kernels.
 * @param input_size The length of the input.
 * @param kernel_size The length of the kernel.
 * @param stride The stride.
 * @param padding The padding.
 *
 */
void convolution1d(float *input, float *output, float *kernels, int input_channels, int kernel_channels, int input_size, int kernel_size, int stride, int padding){

    float *input_cuda, *kernel_cuda, *output_cuda;

    int output_size = static_cast<int>((input_size - kernel_size + padding*2) / (stride+1)) + 1;

    hipMalloc(& input_cuda,                    input_channels *  input_size * sizeof(float));
    hipMalloc(&kernel_cuda,  kernel_channels * input_channels * kernel_size * sizeof(float));
    hipMalloc(&output_cuda,  kernel_channels *                  output_size * sizeof(float));

    hipMemcpy( input_cuda,   input,                    input_channels *  input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(kernel_cuda, kernels,  kernel_channels * input_channels * kernel_size * sizeof(float), hipMemcpyHostToDevice);

    convolution1d_channel<<<dim3(output_size, kernel_channels), dim3(kernel_size, input_channels)>>>(
        input_cuda,
        kernel_cuda,
        output_cuda,
        input_channels,
        kernel_channels,
        input_size,
        kernel_size,
        stride+1,
        padding,
        output_size
    );

    hipMemcpy(output, output_cuda, kernel_channels * output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree( input_cuda);
    hipFree(output_cuda);
    hipFree(kernel_cuda);
}