
// author: caleb7023

#include <hip/hip_runtime.h>

// Its 3D!

/**
 * @brief This is the kernel function for the convolution operation.
 *
 * @param input: input image
 * @param kernel: kernel
 * @param output: output image
 * @param ic: input channels
 * @param kc: kernel channels
 * @param isx: input size x
 * @param isy: input size y
 * @param isz: input size z
 * @param ksx: kernel size x
 * @param ksy: kernel size y
 * @param ksz: kernel size z
 * @param osx: output size x
 * @param osy: output size y
 * @param osz: output size z
 * @param sx: stride x
 * @param sy: stride y
 * @param sz: stride z
 * @param px: padding x
 * @param py: padding y
 * @param pz: padding z
 */
__global__ void convolution3d_ch(
    const float *input, const float *kernel, float *output,
    const unsigned int ic , const unsigned int kc ,
    const unsigned int isx, const unsigned int isy, const unsigned int isz,
    const unsigned int ksx, const unsigned int ksy, const unsigned int ksz,
    const unsigned int osx, const unsigned int osy, const unsigned int osz,
    const unsigned int sx , const unsigned int sy , const unsigned int sz ,
    const unsigned int px , const unsigned int py , const unsigned int pz
)
{
    
    const unsigned int kx_ = threadIdx.x/ksx; // kernel x position
    const unsigned int ky_ = threadIdx.x%ksx; // kernel y position
    const unsigned int kz_ = threadIdx.y; // kernel z position
    const unsigned int ic_ = threadIdx.z; // input channel
    const unsigned int ox_ = blockIdx.x/osx; // target x position in the output
    const unsigned int oy_ = blockIdx.x%osx; // target y position in the output
    const unsigned int oz_ = blockIdx.y; // target z position in the output
    const unsigned int kc_ = blockIdx.z; // kernel/output channel

    const int ix = ox_-px + kx_*sx; // input x position
    const int iy = oy_-py + ky_*sy; // input y position
    const int iz = oz_-pz + kz_*sz; // input z position

    if (kc_ < kc  && // kernel channel
        oy_ < osy && // target y position in the input
        ox_ < osx && // target x position in the input
        oz_ < osz && // target z position in the input
        ic_ < ic  && // input channel
        ky_ < ksy && // kernel y position
        kx_ < ksx && // kernel x position
        kz_ < ksz && // kernel z position
        0<=ix && ix<isx && // check if the input x position is valid
        0<=iy && iy<isy && // check if the input y position is valid
        0<=iz && iz<isz)   // check if the input z position is valid
    {
        atomicAdd(
            &output[kc_*osx*osy*osz + ox_*osz*osy + oy_*osx + oz_],
              input[ic_*isx*isy*isz + ix *isz*isy + iy *isx + iz ] * kernel[kc_*ic*ksx*ksy*ksz + ic_*ksx*ksy*ksz + kx_*ksz*ksy + ky_*ksz + kz_]
        );
    }

}



/**
 *
 * @brief Performs a convolution operation on the input object.
 *
 * The input size does not include the channel size
 * The kernel size does not include the channel size too
 *
 *
 * @param input The input object.
 * @param output The output object.
 * @param kernels The convolution kernels.
 * @param input_channels The number of channels in the input object.
 * @param kernel_channels The number of channels in the kernels.
 * @param input_size_x The width of the input object.
 * @param input_size_y The height of the input object.
 * @param input_size_z The depth of the input object.
 * @param kernel_size_x The width of the kernels.
 * @param kernel_size_y The height of the kernels.
 * @param kernel_size_z The depth of the kernels.
 * @param stride_x The stride in the x direction.
 * @param stride_y The stride in the y direction.
 * @param stride_z The stride in the z direction.
 * @param padding_x The padding in the x direction.
 * @param padding_y The padding in the y direction.
 * @param padding_z The padding in the z direction.
 *
 */
extern "C" void convolve3d(
    const float *input, float *output, const float *kernels,
    const unsigned int input_channels, const unsigned int kernel_channels,
    const unsigned int input_size_x  , const unsigned int input_size_y   , const unsigned int input_size_z ,
    const unsigned int kernel_size_x , const unsigned int kernel_size_y  , const unsigned int kernel_size_z,
    const unsigned int output_size_x , const unsigned int output_size_y  , const unsigned int output_size_z,
    const unsigned int stride_x      , const unsigned int stride_y       , const unsigned int stride_z     ,
    const unsigned int padding_x     , const unsigned int padding_y      , const unsigned int padding_z
)
{

    float *input_cuda, *kernel_cuda, *output_cuda;

    hipMalloc(& input_cuda,                    input_channels *  input_size_x *  input_size_y *  input_size_y * sizeof(float));
    hipMalloc(&kernel_cuda,  kernel_channels * input_channels * kernel_size_x * kernel_size_y * kernel_size_y * sizeof(float));
    hipMalloc(&output_cuda,  kernel_channels *                  output_size_x * output_size_y * output_size_z * sizeof(float));

    hipMemcpy( input_cuda,   input,                    input_channels *  input_size_x *  input_size_y *  input_size_z * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(kernel_cuda, kernels,  kernel_channels * input_channels * kernel_size_x * kernel_size_y * kernel_size_z * sizeof(float), hipMemcpyHostToDevice);

    convolution3d_ch<<<dim3(output_size_x*output_size_y, output_size_z, kernel_channels), dim3(kernel_size_x*kernel_size_y, kernel_size_z, input_channels)>>>(
        input_cuda    , kernel_cuda    , output_cuda,
        input_channels, kernel_channels,
        input_size_x  , input_size_y   , input_size_z ,
        kernel_size_x , kernel_size_y  , kernel_size_z,
        output_size_x , output_size_y  , output_size_z,
        stride_x      , stride_y       , stride_z     ,
        padding_x     , padding_y      , padding_z
    );

    hipMemcpy(output, output_cuda, kernel_channels * output_size_x * output_size_y * output_size_z * sizeof(float), hipMemcpyDeviceToHost);

    hipFree( input_cuda);
    hipFree(output_cuda);
    hipFree(kernel_cuda);

}