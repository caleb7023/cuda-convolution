
// author: caleb7023

#include <hip/hip_runtime.h>

// Its 3D!

/**
 * @brief This is the kernel function for the convolution operation.
 *
 * @param input: input image
 * @param kernel: kernel
 * @param output: output image
 * @param ic: input channels
 * @param kc: kernel channels
 * @param isx: input size x
 * @param isy: input size y
 * @param isz: input size z
 * @param ksx: kernel size x
 * @param ksy: kernel size y
 * @param ksz: kernel size z
 * @param osx: output size x
 * @param osy: output size y
 * @param osz: output size z
 * @param sx: stride x
 * @param sy: stride y
 * @param sz: stride z
 * @param px: padding x
 * @param py: padding y
 * @param pz: padding z
 */
__global__ void convolution3d_channel(
    float *input, float *kernel, float *output,
    int ic , int kc ,
    int isx, int isy, int isz,
    int ksx, int ksy, int ksz,
    int osx, int osy, int osz,
    int sx , int sy , int sz ,
    int px , int py , int pz
)
{
    
    int kx_ = threadIdx.x/ksx; // kernel x position
    int ky_ = threadIdx.x%ksx; // kernel y position
    int kz_ = threadIdx.y; // kernel z position
    int ic_ = threadIdx.z; // input channel
    int ox_ = blockIdx.x/osx; // target x position in the output
    int oy_ = blockIdx.x%osx; // target y position in the output
    int oz_ = blockIdx.y; // target z position in the output
    int kc_ = blockIdx.z; // kernel/output channel

    int ix = ox_-px + kx_*sx; // input x position
    int iy = oy_-py + ky_*sy; // input y position
    int iz = oz_-pz + kz_*sz; // input z position

    if (kc_ < kc  && // kernel channel
        oy_ < osy && // target y position in the input
        ox_ < osx && // target x position in the input
        oz_ < osz && // target z position in the input
        ic_ < ic  && // input channel
        ky_ < ksy && // kernel y position
        kx_ < ksx && // kernel x position
        kz_ < ksz && // kernel z position
        0<=ix && ix<isx && // check if the input x position is valid
        0<=iy && iy<isy && // check if the input y position is valid
        0<=iz && iz<isz)   // check if the input z position is valid
    {
        atomicAdd(
            &output[kc_*osx*osy*osz + ox_*osx*osy + oy_*osy + oz_],
              input[ic_*isx*isy*isz + ix *isx*isy + iy *isy + iz ] * kernel[kc_*ic*ksx*ksy*ksz + ic_*ksx*ksy*ksz + kx_*ksx*ksy + ky_*ksy + kz_]
        );
    }

}



/**
 *
 * @brief Performs a convolution operation on the input object.
 *
 * The input size does not include the channel size
 * The kernel size does not include the channel size too
 *
 *
 * @param input The input object.
 * @param output The output object.
 * @param kernels The convolution kernels.
 * @param input_channels The number of channels in the input object.
 * @param kernel_channels The number of channels in the kernels.
 * @param input_size_x The width of the input object.
 * @param input_size_y The height of the input object.
 * @param input_size_z The depth of the input object.
 * @param kernel_size_x The width of the kernels.
 * @param kernel_size_y The height of the kernels.
 * @param kernel_size_z The depth of the kernels.
 * @param stride_x The stride in the x direction.
 * @param stride_y The stride in the y direction.
 * @param stride_z The stride in the z direction.
 * @param padding_x The padding in the x direction.
 * @param padding_y The padding in the y direction.
 * @param padding_z The padding in the z direction.
 *
 */
extern "C" void convolve3d(
    float *input, float *output, float *kernels,
    int input_channels, int kernel_channels,
    int input_size_x  , int input_size_y   , int input_size_z ,
    int kernel_size_x , int kernel_size_y  , int kernel_size_z,
    int output_size_x , int output_size_y  , int output_size_z,
    int stride_x      , int stride_y       , int stride_z     ,
    int padding_x     , int padding_y      , int padding_z
)
{

    float *input_cuda, *kernel_cuda, *output_cuda;

    hipMalloc(& input_cuda,                    input_channels *  input_size_x *  input_size_y *  input_size_y * sizeof(float));
    hipMalloc(&kernel_cuda,  kernel_channels * input_channels * kernel_size_x * kernel_size_y * kernel_size_y * sizeof(float));
    hipMalloc(&output_cuda,  kernel_channels *                  output_size_x * output_size_y * output_size_z * sizeof(float));

    hipMemcpy( input_cuda,   input,                    input_channels *  input_size_x *  input_size_y *  input_size_z * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(kernel_cuda, kernels,  kernel_channels * input_channels * kernel_size_x * kernel_size_y * kernel_size_z * sizeof(float), hipMemcpyHostToDevice);

    convolution3d_channel<<<dim3(output_size_x*output_size_y, output_size_z, kernel_channels), dim3(kernel_size_x*kernel_size_y, kernel_size_z, input_channels)>>>(
        input_cuda    , kernel_cuda    , output_cuda,
        input_channels, kernel_channels,
        input_size_x  , input_size_y   , input_size_z ,
        kernel_size_x , kernel_size_y  , kernel_size_z,
        output_size_x , output_size_y  , output_size_z,
        stride_x+1    , stride_y+1     , stride_z+1   ,
        padding_x     , padding_y      , padding_z
    );

    hipMemcpy(output, output_cuda, kernel_channels * output_size_x * output_size_y * output_size_z * sizeof(float), hipMemcpyDeviceToHost);

    hipFree( input_cuda);
    hipFree(output_cuda);
    hipFree(kernel_cuda);

}